﻿#include <iostream>

#define THREAD_PER_BLOCK_X  32
#define THREAD_PER_BLOCK_Y  32

#define MATRIX_BLOCK_WIDTH (1024 * 16)
#define MATRIX_BLOCK_HEIGHT (1024 * 16)

#include <iostream>
#include <chrono>
#include <utility>

#include "helper_image.h"

#include "grey.cuh"
#include "RGB.cuh"

#include "hip/hip_runtime.h"
#include ""


template<typename F, typename... Args>
double measureTime(F func, Args&&... args);
void CmpResult(unsigned char* a, unsigned char* b, int size, const std::string& a_string, const std::string& b_string);
void Host_Filter(unsigned char* data, unsigned char* res, unsigned int w, unsigned int h, int channels);


template<typename F, typename... Args>
double measureTime(F func, Args&&... args) {
#define duration(a) std::chrono::duration_cast<std::chrono::microseconds>(a).count()
#define timeNow() std::chrono::high_resolution_clock::now()
    typedef std::chrono::high_resolution_clock::time_point TimeVar;
    TimeVar t1 = timeNow();
    func(std::forward<Args>(args)...);
    TimeVar t2 = timeNow();
    return duration(t2 - t1) / 1000.;
}

void tobin(unsigned char a)
{
    char bin[8];
    for (int i = 0; i < 8; i++)
    {
        bin[i] = a & (1 << (7 - i));
    }
    for (int i = 0; i < 8; i++)
    {
        std::cout << (bool)bin[i];

    }
}

void Host_Filter(unsigned char* data, unsigned char* res, unsigned int width, unsigned int height, int channel)
{
    for (int i = 0; i < height; i++)
    {
        unsigned char* curLine = data + i * width * channel;
        unsigned char* nextLine = data + (i + 1) * width * channel;
        unsigned char* prevLine = data + (i - 1) * width * channel;
        unsigned char* resLine = res + i * width * channel;
        for (int j = 0; j < width * channel; j++)
        {
            int pixel = 0;
            if (i == 0 || i == height - 1) // первая / последняя строчка
            {
                if (i == 0) // первая
                {
                    if (j - channel < 0)    // первый пиксель (+)
                    {
                        pixel = 3 * curLine[j] - 8 * curLine[j] + 2 * curLine[j + channel]
                            + 2 * nextLine[j] + nextLine[j + channel];
                    }
                    if (j + channel >= width * channel) // последний пиксель первой строки (+)
                    {
                        pixel = 3 * curLine[j] - 8 * curLine[j] + 2 * curLine[j - channel]
                            + 2 * nextLine[j] + nextLine[j - channel];
                    }
                    if (j - channel >= 0 && j + channel < width * channel)  // между первым и последним (+)
                    {
                        pixel = 2 * curLine[j - channel] + curLine[j] - 8 * curLine[j] + 2 *  curLine[j + channel]
                            + nextLine[j - channel] + nextLine[j] + nextLine[j + channel];
                    }
                }
                if (i == height - 1)    // последняя
                {
                    if (j - channel < 0)    //крайний левый пиксель (+)
                    {
                        pixel = 2 * prevLine[j] + prevLine[j + channel]
                            - 8 * curLine[j] + 3 * curLine[j] + 2 * curLine[j + channel];
                    }
                    if (j + channel >= width * channel) // крайний правый пиксель (+)
                    {
                        pixel = prevLine[j - channel] + 2 * prevLine[j]
                            + 2 * curLine[j - channel] - 8 * curLine[j] + 3 * curLine[j];
                    }
                    if (j - channel >= 0 && j + channel < width * channel)  // между крайними (+)
                    {
                        pixel = prevLine[j - channel] + prevLine[j] + prevLine[j + channel]
                            + 2 * curLine[j - channel] - 8 * curLine[j] + curLine[j] + 2 * curLine[j + channel];
                    }
                }
            }
            else
            {
                if (j - channel < 0)    // первый столбец (+)
                {
                    pixel = 2 * prevLine[j] + prevLine[j + channel]
                        + curLine[j] - 8 * curLine[j] + curLine[j + channel]
                        + 2 * nextLine[j] + nextLine[j + channel];
                }
                else
                {
                    if (j + channel >= width * channel) // последний столбец (+)
                    {
                        pixel = prevLine[j - channel] + 2 * prevLine[j]
                            + curLine[j - channel] - 8 * curLine[j] + curLine[j]
                            + nextLine[j - channel] + 2 * nextLine[j];
                    }
                    else    // все остальные нормальные пиксели, а не вот это вот
                    {
                        pixel = prevLine[j - channel] + prevLine[j] + prevLine[j + channel]
                            + curLine[j - channel] - 8 * curLine[j] + curLine[j + channel]
                            + nextLine[j - channel] + nextLine[j] + nextLine[j + channel];
                    }
                }
            }
            pixel = pixel > 255 ? 255 : pixel;
            pixel = pixel < 0 ? 0 : pixel;
            resLine[j] = (unsigned char)pixel;
        }
    }
}

void CmpResult(unsigned char* a, unsigned char* b, int size, const std::string& a_string, const std::string& b_string) {
    int cmp_val = memcmp(a, b, size);
    std::cout << a_string << " ";
    if (cmp_val == 0) {
        std::cout << "==";
    }
    else {
        std::cout << "!=";
    }
    std::cout << " " << b_string << '\n';
}

int main() {
    std::string dir = R"(C:\Users\danilapoddubny\source\repos\ImageProcessing\ImageProcessing\)";
    std::string name = "ontos";

    unsigned char* data = nullptr;
    unsigned int w = 0;
    unsigned int h = 0;
    unsigned int channels = 0;

    __loadPPM((dir + name + ".pgm").c_str(), &data, &w, &h, &channels);
    unsigned char* res = (unsigned char*)malloc(w * h * sizeof(unsigned char) * channels);
    unsigned char* res_dev = (unsigned char*)malloc(w * h * sizeof(unsigned char) * channels);

    memset(res, 0, w * h * sizeof(unsigned char) * channels);
    double host_time = measureTime(Host_Filter, data, res, w, h, channels);
    std::cout << "Host " + name + " (" << w << 'x' << h << "): " << host_time << " ms\n";
    __savePPM((dir + name + "_host.pgm").c_str(), res, w, h, channels);

    double device_opt_time = grey_filter_Optimized(res_dev, data, w, h);
    std::cout << "Device optimized " + name + " (" << w << 'x' << h << "): " << device_opt_time << " ms\n";
    CmpResult(res, res_dev, w * h * channels * sizeof(unsigned char), "Host", "Device opt");
    __savePPM((dir + name + "_device_opt.pgm").c_str(), res_dev, w, h, channels);

    free(data);
    free(res);
    free(res_dev);

    data = nullptr;
    w = 0;
    h = 0;
    channels = 0;

    __loadPPM((dir + name + ".ppm").c_str(), &data, &w, &h, &channels);

    res = (unsigned char*)malloc(w * h * sizeof(unsigned char) * channels);
    res_dev = (unsigned char*)malloc(w * h * sizeof(unsigned char) * channels);

    memset(res, 0, w * h * sizeof(unsigned char) * channels);
    host_time = measureTime(Host_Filter, data, res, w, h, channels);
    std::cout << "Host " + name + " (" << w << 'x' << h << "): " << host_time << " ms\n";
    __savePPM((dir + name + "_host.ppm").c_str(), res, w, h, channels);

    memset(res_dev, 0, w * h * sizeof(unsigned char) * channels);
    device_opt_time = RGB_filter_Optimized(res_dev, data, w, h);
    std::cout << "Device optimized " + name + " (" << w << 'x' << h << "): " << device_opt_time << " ms\n";
    __savePPM((dir + name + "_device_opt.ppm").c_str(), res_dev, w, h, channels);
    CmpResult(res_dev, res, w * h * channels * sizeof(unsigned char), "Host", "Device opt");

    free(data);
    free(res);
    free(res_dev);

    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceReset failed!\n";
        return 1;
    }

	return 0;
}