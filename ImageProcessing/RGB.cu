#include "hip/hip_runtime.h"
//
// Created by maria on 20.04.2022.
//

#include "RGB.cuh"
#define THREAD_PER_BLOCK_X  32
#define THREAD_PER_BLOCK_Y  32

#define MATRIX_BLOCK_WIDTH (1024 * 16)
#define MATRIX_BLOCK_HEIGHT (1024 * 16)

#include "hip/hip_runtime.h"
#include ""

#include <iostream>


__global__ void RGB_kernel(unsigned char* dst, unsigned char* src, int height, int width, int s_pitch, int d_pitch) {

    int thread_x = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_y = blockIdx.y * blockDim.y + threadIdx.y;
    int dst_center = d_pitch * thread_y + thread_x;
    int src_center = s_pitch * (thread_y + 1) + thread_x + 3;
    int src_low = s_pitch * (thread_y + 2) + thread_x + 3;
    int src_high = s_pitch * (thread_y)+thread_x + 3;
    int elem = 0;
    if (thread_x < width * 3 && thread_y < height) {
        elem = src[src_high - 3] + src[src_high] + src[src_high + 3]
            + src[src_center - 3] - 8 * src[src_center] + src[src_center + 3]
            + src[src_low - 3] + src[src_low] + src[src_low + 3];
        elem = elem > 255 ? 255 : elem;
        elem = elem < 0 ? 0 : elem;
        dst[dst_center] = (unsigned char)elem;
    }

}


__global__ void RGB_kernel_optimized(uint32_t* dst, uint32_t* src, int height, int width, int s_pitch, int d_pitch) {
    __shared__ unsigned char mem[3 * (4 * THREAD_PER_BLOCK_X + 8) * (THREAD_PER_BLOCK_Y + 2)];


    int thread_x = 3 * blockIdx.x * blockDim.x + threadIdx.x;
    int thread_y = blockIdx.y * blockDim.y + threadIdx.y;
    int dst_center = d_pitch * thread_y + thread_x;
    int src_center = s_pitch * (thread_y + 1) + thread_x + 128 / 4;
    int src_low = s_pitch * (thread_y + 2) + thread_x + 128 / 4;
    int src_high = s_pitch * (thread_y)+thread_x + 128 / 4;

    int mem_center = (threadIdx.y + 1) * 3 * (4 * THREAD_PER_BLOCK_X + 8) + threadIdx.x * 4 + 12;
    int mem_low = (threadIdx.y + 2) * 3 * (4 * THREAD_PER_BLOCK_X + 8) + threadIdx.x * 4 + 12;
    int mem_high = threadIdx.y * 3 * (4 * THREAD_PER_BLOCK_X + 8) + threadIdx.x * 4 + 12;

    int mem32_center = mem_center / 4;
    int mem32_low = mem_low / 4;
    int mem32_high = mem_high / 4;

    uint32_t* mem32 = (uint32_t*)mem;

    if (thread_x * 4 < 12 + width * 3 && thread_y <= height) {
        mem32[mem32_center] = src[src_center];
        mem32[mem32_center + THREAD_PER_BLOCK_X] = src[src_center + THREAD_PER_BLOCK_X];
        mem32[mem32_center + 2 * THREAD_PER_BLOCK_X] = src[src_center + 2 * THREAD_PER_BLOCK_X];

        __syncthreads();

        if (threadIdx.x == 0) {
            mem32[mem32_center - 1] = src[src_center - 1];
            if (threadIdx.y == blockDim.y - 1)
            {
                mem32[mem32_low - 1] = src[src_low - 1];
                mem32[mem32_low - 1 + 2 * THREAD_PER_BLOCK_X] = src[src_low - 1 + 2 * THREAD_PER_BLOCK_X];

            }
        }
        if (threadIdx.x == blockDim.x - 1) {
            mem32[mem32_center + 1 + 2 * THREAD_PER_BLOCK_X] = src[src_center + 1 + 2 * THREAD_PER_BLOCK_X];
            if (threadIdx.y == 0)
            {
                mem32[mem32_high + 1 + 2 * THREAD_PER_BLOCK_X] = src[src_high + 1 + 2 * THREAD_PER_BLOCK_X];
            }
        }
        if (threadIdx.y == 0) {
            mem32[mem32_high] = src[src_high];
            mem32[mem32_high + THREAD_PER_BLOCK_X] = src[src_high + THREAD_PER_BLOCK_X];
            mem32[mem32_high + 2 * THREAD_PER_BLOCK_X] = src[src_high + 2 * THREAD_PER_BLOCK_X];
            if (threadIdx.x == 0)
            {
                mem32[mem32_high - 1] = src[src_high - 1];
            }
        }

        if (threadIdx.y == blockDim.y - 1) {
            mem32[mem32_low] = src[src_low];
            mem32[mem32_low + THREAD_PER_BLOCK_X] = src[src_low + THREAD_PER_BLOCK_X];
            mem32[mem32_low + 2 * THREAD_PER_BLOCK_X] = src[src_low + 2 * THREAD_PER_BLOCK_X];
            if (threadIdx.x == blockDim.x - 1) {
                mem32[mem32_low + 1 + 2 * THREAD_PER_BLOCK_X] = src[src_low + 1 + 2 * THREAD_PER_BLOCK_X];
            }
        }

        __syncthreads();


        for (int i = 0; i < 3; i++) {
            uint32_t res = 0;
            for (int j = 0; j < 4; j++) {
                int32_t elem = 0;
                elem = -8 * mem[mem_center + j + 4 * i * THREAD_PER_BLOCK_X]
                    + mem[mem_high - 3 + j + 4 * i * THREAD_PER_BLOCK_X] + mem[mem_high + j + 4 * i * THREAD_PER_BLOCK_X] + mem[mem_high + 3 + j + 4 * i * THREAD_PER_BLOCK_X]
                    + mem[mem_center - 3 + j + 4 * i * THREAD_PER_BLOCK_X] + mem[mem_center + 3 + j + 4 * i * THREAD_PER_BLOCK_X]
                    + mem[mem_low - 3 + j + 4 * i * THREAD_PER_BLOCK_X] + mem[mem_low + j + 4 * i * THREAD_PER_BLOCK_X] + mem[mem_low + 3 + j + 4 * i * THREAD_PER_BLOCK_X];
                elem = elem > 255 ? 255 : elem;
                elem = elem < 0 ? 0 : elem;
                res |= ((elem & 0xFF) << (8 * j));
            }
            dst[dst_center + i * THREAD_PER_BLOCK_X] = res;
        }

    }

}


double RGB_filter(unsigned char* host_dst, unsigned char* host_src, int width, int height) {
    hipError_t cudaStatus = hipSuccess;
    unsigned char* dev_src = nullptr;
    unsigned char* dev_dst = nullptr;
    size_t s_pitch = 0;
    size_t d_pitch = 0;
    int channels = 3;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMallocPitch((void**)&dev_src, &s_pitch, channels * (2 + MATRIX_BLOCK_WIDTH) * sizeof(unsigned char), 2 + MATRIX_BLOCK_HEIGHT);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMallocPitch failed!\n";
        return -1;
    }

    cudaStatus = hipMallocPitch((void**)&dev_dst, &d_pitch, channels * MATRIX_BLOCK_WIDTH * sizeof(unsigned char), MATRIX_BLOCK_HEIGHT);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMallocPitch failed!\n";
        hipFree(dev_src);
        return -1;
    }

    hipEventRecord(start, 0);

    int max_i = (height + MATRIX_BLOCK_HEIGHT - 1) / MATRIX_BLOCK_HEIGHT;
    int max_j = (width + MATRIX_BLOCK_WIDTH - 1) / MATRIX_BLOCK_WIDTH;
    for (int i = 0; i < max_i; i++) {
        for (int j = 0; j < max_j; j++) {

            int block_width = MATRIX_BLOCK_WIDTH;
            int block_height = MATRIX_BLOCK_HEIGHT;

            if (block_width * j + block_width > width) {
                block_width = width - block_width * j;
            }

            if (block_height * i + block_height > height) {
                block_height = height - block_height * i;
            }
            int host_src_index = i * MATRIX_BLOCK_HEIGHT * width * channels + MATRIX_BLOCK_WIDTH * j * channels;
            int dev_src_column_index = 1 * channels;
            int dev_src_row_index = (int)s_pitch;
            int read_block_width = block_width * channels;
            int read_block_height = block_height;
            if (j != 0) {
                dev_src_column_index -= channels;
                host_src_index -= channels;
                read_block_width += channels;
            }
            if (i != 0) {
                dev_src_row_index -= (int)s_pitch;
                host_src_index -= width * channels;
                read_block_height++;
            }
            if (i != max_i - 1) {
                read_block_height++;
            }
            if (j != max_j - 1) {
                read_block_width += channels;
            }

            // Copy input vectors from host memory to GPU buffers.
            cudaStatus = hipMemcpy2DAsync(dev_src + dev_src_column_index + dev_src_row_index,
                s_pitch,
                host_src + host_src_index,
                width * channels * sizeof(char),
                read_block_width * sizeof(char),
                read_block_height,
                hipMemcpyHostToDevice);
            if (cudaStatus != hipSuccess) {
                std::cerr << "cudaMemcpy2DAsync1 failed!\n";
                hipFree(dev_src);
                hipFree(dev_dst);
                return -1;
            }
            if (i == 0) {
                cudaStatus = hipMemcpy2DAsync(dev_src + dev_src_column_index,
                    s_pitch,
                    host_src + host_src_index,
                    width * channels * sizeof(char),
                    read_block_width * sizeof(char),
                    1,
                    hipMemcpyHostToDevice);
                if (cudaStatus != hipSuccess) {
                    std::cerr << "cudaMemcpy2DAsync2 failed!\n";
                    hipFree(dev_src);
                    hipFree(dev_dst);
                    return -1;
                }
            }
            if (i == max_i - 1) {
                if (i != 0) {
                    host_src_index += width * channels;
                }
                cudaStatus = hipMemcpy2DAsync(dev_src + s_pitch * (block_height + 1) + dev_src_column_index,
                    s_pitch,
                    host_src + host_src_index + (block_height - 1) * width * channels,
                    width * channels * sizeof(char),
                    read_block_width * sizeof(char),
                    1,
                    hipMemcpyHostToDevice);
                if (cudaStatus != hipSuccess) {
                    std::cerr << "cudaMemcpy2DAsync3 failed!\n";
                    hipFree(dev_src);
                    hipFree(dev_dst);
                    return -1;
                }
            }
            if (j == 0) {
                for (int k = 0; k < block_height + 2; k++) {
                    cudaStatus = hipMemcpyAsync(dev_src + s_pitch * k,
                        dev_src + s_pitch * k + channels,
                        channels,
                        hipMemcpyDeviceToDevice);
                    if (cudaStatus != hipSuccess) {
                        std::cerr << "cudaMemcpyAsync4 failed!\n";
                        hipFree(dev_src);
                        hipFree(dev_dst);
                        return -1;
                    }
                }
            }

            if (j == max_j - 1) {
                for (int k = 0; k < block_height + 2; k++) {

                    cudaStatus = hipMemcpyAsync(dev_src + s_pitch * k + channels + block_width * channels,
                        dev_src + s_pitch * k + block_width * channels,
                        channels,
                        hipMemcpyDeviceToDevice);
                    if (cudaStatus != hipSuccess) {
                        std::cerr << "cudaMemcpyAsync5 failed!\n";
                        hipFree(dev_src);
                        hipFree(dev_dst);
                        return -1;
                    }

                }
            }

            dim3 block(THREAD_PER_BLOCK_X, THREAD_PER_BLOCK_Y);
            dim3 grid((block_width * channels + THREAD_PER_BLOCK_X - 1) / THREAD_PER_BLOCK_X, (block_height + THREAD_PER_BLOCK_Y - 1) / THREAD_PER_BLOCK_Y);
            // Launch a kernel on the GPU with one thread for each element.
            RGB_kernel << <grid, block >> > (dev_dst, dev_src, block_height, block_width, (int)s_pitch, (int)d_pitch);

            // Check for any errors launching the kernel
            cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess) {
                std::cerr << "grey_Kernel launch failed: " << hipGetErrorString(cudaStatus) << "\n";
                hipFree(dev_src);
                hipFree(dev_dst);
                return -1;
            }

            // Copy output vector from GPU buffer to host memory.
            cudaStatus = hipMemcpy2DAsync(host_dst + i * MATRIX_BLOCK_HEIGHT * width * channels + MATRIX_BLOCK_WIDTH * j * channels,
                width * channels * sizeof(char),
                dev_dst,
                d_pitch,
                block_width * channels * sizeof(char),
                block_height,
                hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                std::cerr << "cudaMemcpy2DAsync6 failed!\n";
                hipFree(dev_src);
                hipFree(dev_dst);
                return -1;
            }
        }
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipFree(dev_src);
    hipFree(dev_dst);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsedTime;
}


double RGB_filter_Optimized(unsigned char* host_dst, unsigned char* host_src, int width, int height) {
    hipError_t cudaStatus = hipSuccess;
    unsigned char* dev_src = nullptr;
    unsigned char* dev_dst = nullptr;
    size_t s_pitch = 0;
    size_t d_pitch = 0;
    int channels = 3;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMallocPitch((void**)&dev_src, &s_pitch, channels * (2 + MATRIX_BLOCK_WIDTH) * sizeof(unsigned char), 2 + MATRIX_BLOCK_HEIGHT);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMallocPitch failed!\n";
        return -1;
    }

    cudaStatus = hipMallocPitch((void**)&dev_dst, &d_pitch, channels * MATRIX_BLOCK_WIDTH * sizeof(unsigned char), MATRIX_BLOCK_HEIGHT);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMallocPitch failed!\n";
        hipFree(dev_src);
        return -1;
    }

    hipEventRecord(start, 0);

    int max_i = (height + MATRIX_BLOCK_HEIGHT - 1) / MATRIX_BLOCK_HEIGHT;
    int max_j = (width + MATRIX_BLOCK_WIDTH - 1) / MATRIX_BLOCK_WIDTH;
    for (int i = 0; i < max_i; i++) {
        for (int j = 0; j < max_j; j++) {

            int block_width = MATRIX_BLOCK_WIDTH;
            int block_height = MATRIX_BLOCK_HEIGHT;

            if (block_width * j + block_width > width) {
                block_width = width - block_width * j;
            }

            if (block_height * i + block_height > height) {
                block_height = height - block_height * i;
            }
            int host_src_index = i * MATRIX_BLOCK_HEIGHT * width * channels + MATRIX_BLOCK_WIDTH * j * channels;
            int dev_src_column_index = 128;
            int dev_src_row_index = (int)s_pitch;
            int read_block_width = block_width * channels;
            int read_block_height = block_height;
            if (j != 0) {
                dev_src_column_index -= channels;
                host_src_index -= channels;
                read_block_width += channels;
            }
            if (i != 0) {
                dev_src_row_index -= (int)s_pitch;
                host_src_index -= width * channels;
                read_block_height++;
            }
            if (i != max_i - 1) {
                read_block_height++;
            }
            if (j != max_j - 1) {
                read_block_width += channels;
            }

            // Copy input vectors from host memory to GPU buffers.
            cudaStatus = hipMemcpy2DAsync(dev_src + dev_src_column_index + dev_src_row_index,
                s_pitch,
                host_src + host_src_index,
                width * channels * sizeof(char),
                read_block_width * sizeof(char),
                read_block_height,
                hipMemcpyHostToDevice);
            if (cudaStatus != hipSuccess) {
                std::cerr << "cudaMemcpy2DAsync1 failed!\n";
                hipFree(dev_src);
                hipFree(dev_dst);
                return -1;
            }
            if (i == 0) {
                cudaStatus = hipMemcpy2DAsync(dev_src + dev_src_column_index,
                    s_pitch,
                    host_src + host_src_index,
                    width * channels * sizeof(char),
                    read_block_width * sizeof(char),
                    1,
                    hipMemcpyHostToDevice);
                if (cudaStatus != hipSuccess) {
                    std::cerr << "cudaMemcpy2DAsync2 failed!\n";
                    hipFree(dev_src);
                    hipFree(dev_dst);
                    return -1;
                }
            }
            if (i == max_i - 1) {
                if (i != 0) {
                    host_src_index += width * channels;
                }
                cudaStatus = hipMemcpy2DAsync(dev_src + s_pitch * (block_height + 1) + dev_src_column_index,
                    s_pitch,
                    host_src + host_src_index + (block_height - 1) * width * channels,
                    width * channels * sizeof(char),
                    read_block_width * sizeof(char),
                    1,
                    hipMemcpyHostToDevice);
                if (cudaStatus != hipSuccess) {
                    std::cerr << "cudaMemcpy2DAsync3 failed!\n";
                    hipFree(dev_src);
                    hipFree(dev_dst);
                    return -1;
                }
            }
            if (j == 0) {
                for (int k = 0; k < block_height + 2; k++) {
                    cudaStatus = hipMemcpyAsync(dev_src + s_pitch * k + 125,
                        dev_src + s_pitch * k + 128,
                        channels,
                        hipMemcpyDeviceToDevice);
                    if (cudaStatus != hipSuccess) {
                        std::cerr << "cudaMemcpyAsync4 failed!\n";
                        hipFree(dev_src);
                        hipFree(dev_dst);
                        return -1;
                    }
                }
            }

            if (j == max_j - 1) {
                for (int k = 0; k < block_height + 2; k++) {

                    cudaStatus = hipMemcpyAsync(dev_src + s_pitch * k + 128 + block_width * channels,
                        dev_src + s_pitch * k + block_width * channels + 125,
                        channels,
                        hipMemcpyDeviceToDevice);
                    if (cudaStatus != hipSuccess) {
                        std::cerr << "cudaMemcpyAsync5 failed!\n";
                        hipFree(dev_src);
                        hipFree(dev_dst);
                        return -1;
                    }

                }
            }

            dim3 block(THREAD_PER_BLOCK_X, THREAD_PER_BLOCK_Y);
            dim3 grid(((block_width + 3) / 4 + THREAD_PER_BLOCK_X - 1) / THREAD_PER_BLOCK_X, (block_height + THREAD_PER_BLOCK_Y - 1) / THREAD_PER_BLOCK_Y);
            // Launch a kernel on the GPU with one thread for each element.
            RGB_kernel_optimized << <grid, block >> > ((uint32_t*)dev_dst, (uint32_t*)dev_src, block_height,
                block_width, (int)s_pitch / sizeof(uint32_t),
                (int)d_pitch / sizeof(uint32_t));

            // Check for any errors launching the kernel
            cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess) {
                std::cerr << "grey_Kernel launch failed: " << hipGetErrorString(cudaStatus) << "\n";
                hipFree(dev_src);
                hipFree(dev_dst);
                return -1;
            }

            // Copy output vector from GPU buffer to host memory.
            cudaStatus = hipMemcpy2DAsync(host_dst + i * MATRIX_BLOCK_HEIGHT * width * channels + MATRIX_BLOCK_WIDTH * j * channels,
                width * channels * sizeof(char),
                dev_dst,
                d_pitch,
                block_width * channels * sizeof(char),
                block_height,
                hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                std::cerr << "cudaMemcpy2DAsync6 failed!\n";
                hipFree(dev_src);
                hipFree(dev_dst);
                return -1;
            }
        }
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipFree(dev_src);
    hipFree(dev_dst);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsedTime;
}




